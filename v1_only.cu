#include "hip/hip_runtime.h"
#include <cassert>
#include <cmath>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <string>
#include <iomanip>

constexpr int BlockRowSize = 4; // rows per block
constexpr int BlockColSize = 4;
using FP = float;

// CUDA error checking function
inline static hipError_t checkCuda(hipError_t result, const char* srcStr) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error (src: %s): %s\n", srcStr, hipGetErrorString(result));
        assert(result == hipSuccess);
    }
    return result;
}

struct TestData {
    int seq_len;
    int emb_size;
    std::vector<std::vector<float>> data;
};

std::vector<TestData> loadData(const std::string& directory,
    const std::vector<int>& seq_lengths = { 4, 8, 16, 32 },
    const std::vector<int>& emb_sizes = { 64, 128, 256, 512, 1024 }) {
    std::vector<TestData> dataset;
    for (int seq_len : seq_lengths) {
        for (int emb_size : emb_sizes) {
            std::stringstream filename;
            filename << directory << "/flash_attention_seq" << seq_len << "_emb" << emb_size << ".txt";

            std::ifstream file(filename.str());
            if (!file.is_open()) {
                std::cerr << "Failed to open file: " << filename.str() << std::endl;
                continue;
            }

            TestData test_data{ seq_len, emb_size, {} };
            std::string line;
            std::getline(file, line);  // Skip header
            while (std::getline(file, line)) {
                std::istringstream ss(line);
                test_data.data.push_back({ std::istream_iterator<float>(ss), std::istream_iterator<float>() });
            }
            dataset.push_back(std::move(test_data));
        }
    }
    return dataset;
}

__global__ void flash_attention_v1_kernel(FP* Q, FP* K, FP* V, FP* O, int seqLen, int embDim, FP scaleFactor) {
    extern __shared__ FP sharedMem[]; // shared memory
    FP* sharedQ = sharedMem; // shared memory for Q
    FP* sharedK = sharedQ + BlockRowSize * embDim; // shared memory for K
    FP* sharedV = sharedK + BlockColSize * embDim; // shared memory for V

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int rowIdx = blockIdx.y * blockDim.y + ty;

    if (rowIdx >= seqLen) return;

    // initialize shared memory for Q
    for (int i = tx; i < embDim; i += blockDim.x) {
        sharedQ[ty * embDim + i] = Q[rowIdx * embDim + i];
    }
    __syncthreads();

    FP localMax = -INFINITY;
    FP localDenom = 0.0f;
    FP partialSum = 0.0f;

    for (int k = 0; k < seqLen; k += BlockColSize) {
        if (k + tx < seqLen) {
            for (int i = 0; i < embDim; ++i) {
                sharedK[tx * embDim + i] = K[(k + tx) * embDim + i];
                sharedV[tx * embDim + i] = V[(k + tx) * embDim + i];
            }
        }
        __syncthreads();

        // compute QK
        FP qkSum = 0.0f;
        for (int i = 0; i < embDim; ++i) {
            qkSum += sharedQ[ty * embDim + i] * sharedK[tx * embDim + i];
        }
        qkSum *= scaleFactor;
        localMax = max(localMax, qkSum);
        FP expVal = exp(qkSum);
        localDenom += expVal;

        for (int i = 0; i < embDim; ++i) {
            partialSum += expVal * sharedV[tx * embDim + i];
        }
        __syncthreads();
    }

    // write final result back to global memory
    for (int i = tx; i < embDim; i += blockDim.x) {
        O[rowIdx * embDim + i] = partialSum / localDenom;
    }
}

void flash_attention_v1_cuda(FP* Q, FP* K, FP* V, FP* O, int seqLen, int embDim) {
    FP scaleFactor = 1.f / sqrtf(static_cast<FP>(embDim));
    size_t sharedMemSize = (BlockRowSize * embDim + 2 * BlockColSize * embDim) * sizeof(FP);

    dim3 block(BlockColSize, BlockRowSize);
    dim3 grid(1, (seqLen + BlockRowSize - 1) / BlockRowSize);

    flash_attention_v1_kernel << <grid, block, sharedMemSize >> > (Q, K, V, O, seqLen, embDim, scaleFactor);
    checkCuda(hipDeviceSynchronize(), "flash_attention_v1_cuda");
}

void test_attention(const TestData& test_data) {
    int seq_len = test_data.seq_len;
    int emb_dim = test_data.emb_size;

    std::vector<FP> h_Q(seq_len * emb_dim, 1.0f);
    std::vector<FP> h_K(seq_len * emb_dim, 1.0f);
    std::vector<FP> h_V(seq_len * emb_dim, 1.0f);
    std::vector<FP> h_O(seq_len * emb_dim, 0.0f);

    FP* d_Q, * d_K, * d_V, * d_O;
    checkCuda(hipMalloc(&d_Q, seq_len * emb_dim * sizeof(FP)), "hipMalloc d_Q");
    checkCuda(hipMalloc(&d_K, seq_len * emb_dim * sizeof(FP)), "hipMalloc d_K");
    checkCuda(hipMalloc(&d_V, seq_len * emb_dim * sizeof(FP)), "hipMalloc d_V");
    checkCuda(hipMalloc(&d_O, seq_len * emb_dim * sizeof(FP)), "hipMalloc d_O");

    checkCuda(hipMemcpy(d_Q, h_Q.data(), seq_len * emb_dim * sizeof(FP), hipMemcpyHostToDevice), "hipMemcpy h_Q");
    checkCuda(hipMemcpy(d_K, h_K.data(), seq_len * emb_dim * sizeof(FP), hipMemcpyHostToDevice), "hipMemcpy h_K");
    checkCuda(hipMemcpy(d_V, h_V.data(), seq_len * emb_dim * sizeof(FP), hipMemcpyHostToDevice), "hipMemcpy h_V");

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    flash_attention_v1_cuda(d_Q, d_K, d_V, d_O, seq_len, emb_dim);
    hipEventRecord(stop);

    checkCuda(hipMemcpy(h_O.data(), d_O, seq_len * emb_dim * sizeof(FP), hipMemcpyDeviceToHost), "hipMemcpy h_O");

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    std::cout << std::fixed << std::setprecision(3)
        << "SeqLen: " << seq_len << ", EmbDim: " << emb_dim
        << "  V1 Execution Time: " << milliseconds << " ms\n";

    checkCuda(hipFree(d_Q), "hipFree d_Q");
    checkCuda(hipFree(d_K), "hipFree d_K");
    checkCuda(hipFree(d_V), "hipFree d_V");
    checkCuda(hipFree(d_O), "hipFree d_O");

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    std::string dataset_dir = "C:/Users/PhotonUser/My Files/OneDrive/Files/final/flash_attention_dataset";

    std::vector<TestData> dataset = loadData(dataset_dir);

    for (const auto& test_data : dataset) {
        test_attention(test_data);
    }

    return 0;
}
